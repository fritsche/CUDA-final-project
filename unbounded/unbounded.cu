#include "hip/hip_runtime.h"

#include <stdio.h>

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define POP_SIZE 			32 // the suggested value is 40
#define SOLUTION_SIZE 		2
#define MAX_ITERATIONS 		10
#define FUNCTION 			SPHERE
#define CUDA_MAX_DOUBLE 	8.98847e+307 
#define solution(var)		solutions[(blockIdx.x*SOLUTION_SIZE)+var]
#define objective			objectives[blockIdx.x]

#define LOG_LEVEL			INFO

static void HandleError( hipError_t err,
	const char *file,
	int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ),
			file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__device__ double objective_function (double *solutions) {

	double value;

	#if FUNCTION == SPHERE
		value = 0.0;
		for (int i = 0; i < SOLUTION_SIZE; ++i)
		{
			value += ( solution(i) * solution(i) );
		}
	#endif

	return value;
}

__global__ void pso (hiprandState_t* states, double *solutions, double *objectives) {

	int tid = blockIdx.x;

	double velocity[SOLUTION_SIZE];
	double local_best[SOLUTION_SIZE];
	double local_best_objective;

	for (int var = 0; var < SOLUTION_SIZE; ++var)
	{
		solution(var) = hiprand_uniform(&states[blockIdx.x]); // rand (0.0 .. 1.0)
		velocity[var] = hiprand_uniform(&states[blockIdx.x]); // rand (0.0 .. 1.0)
		local_best[var] = solution(var); // the local_best is initialized with the solution
	}

	// evaluate solution
	objective = objective_function (solutions);

	// __syncthreads();
	// update global best
	// @TODO
	// Probably we need perform a reduce
	// So, the solution and objective need to be global

	for (int it = 0; it < MAX_ITERATIONS; ++it)
	{
		// @TODO
	}
	
	#if LOG_LEVEL == INFO
		printf("%d: [%lf %lf] = %lf\n", tid, solution(0), solution(1), objective);
	#endif
}

/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states) {

  /* we have to initialize the state */
  hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
              blockIdx.x, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              	&states[blockIdx.x]);
}

int main(int argc, char const *argv[])
{
	/* CUDA's random number library uses hiprandState_t to keep track of the seed value
     we will store a random state for every thread  */
	hiprandState_t* states;
  	/* allocate space on the GPU for the random states */
	HANDLE_ERROR( hipMalloc((void**) &states, POP_SIZE * sizeof(hiprandState_t) ) );
  	/* invoke the GPU to initialize all of the random states */
	init<<<POP_SIZE, 1>>>(time(0), states);

	// solutions[POPULATION_SIZE][SOLUTION_SIZE]
	// [p0v0 p0v1 p1v0 p1v1 p2v0 p2v1]
	double *dev_solutions_matrix; 
	double *dev_solutions_objectives;

	HANDLE_ERROR( hipMalloc((void**) &dev_solutions_matrix, SOLUTION_SIZE * POP_SIZE * sizeof(double) ) );
	HANDLE_ERROR( hipMalloc((void**) &dev_solutions_objectives, POP_SIZE * sizeof(double) ) );

	pso<<<POP_SIZE,1>>>(states, dev_solutions_matrix, dev_solutions_objectives);

	// hipDeviceSynchronize is used to allow printf inside device functions
	// http://stackoverflow.com/questions/19193468/why-do-we-need-cudadevicesynchronize-in-kernels-with-device-printf
	hipDeviceSynchronize();
	
	return 0;
}
