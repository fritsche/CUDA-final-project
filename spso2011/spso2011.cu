#include "hip/hip_runtime.h"

#include <stdio.h>

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define POP_SIZE 			32 // the suggested value is 40
#define SOLUTION_SIZE 		2
#define MAX_ITERATIONS 		10
#define FUNCTION 			SPHERE
#define CUDA_MAX_DOUBLE 	8.98847e+307 
#define MIN_VALUE			-100.0
#define MAX_VALUE			100.0

#define solution(var)		solutions[(blockIdx.x*SOLUTION_SIZE)+var]
#define objective			objectives[blockIdx.x]
#define rand(min,max)		(max-min)*hiprand_uniform(&states[blockIdx.x])+min

#define LOG_LEVEL			INFO

static void HandleError( hipError_t err,
	const char *file,
	int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ),
			file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__device__ double objective_function (double *solutions) {

	double value;

	#if FUNCTION == SPHERE
		value = 0.0;
		for (int i = 0; i < SOLUTION_SIZE; ++i)
		{
			value += ( solution(i) * solution(i) );
		}
	#endif

	return value;
}

__global__ void pso (hiprandState_t* states, double *solutions, double *objectives) {

	int tid = blockIdx.x;

	double velocity[SOLUTION_SIZE];
	double local_best[SOLUTION_SIZE]; // personal best
	double local_best_objective;

	// initialization
	for (int var = 0; var < SOLUTION_SIZE; ++var)
	{
		// initialize the swarm
		// http://cs.umw.edu/~finlayson/class/fall14/cpsc425/notes/23-cuda-random.html
		// x_i = U (min_d, max_d)
		solution(var) = rand(MIN_VALUE, MAX_VALUE);
		// v_i = U (mind - x_{i,d} ,maxd - x_{i,d})
		velocity[var] = rand(MIN_VALUE - solution(var), MAX_VALUE - solution(var));
		// p_i = x_i
		local_best[var] = solution(var); // the local_best is initialized with the solution
	}

	// evaluate solution
	objective = objective_function (solutions);

	// __syncthreads();
	// update global best
	// @TODO
	// Probably we need perform a reduce
	// So, the solution and objective need to be global

	for (int it = 0; it < MAX_ITERATIONS; ++it)
	{
		// @TODO
	}
	
	#if LOG_LEVEL == INFO
		printf("%d: x = [%lf %lf] v = [%lf %lf] = %lf\n", tid, solution(0), solution(1), velocity[0], velocity[1], objective);
	#endif
}

/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states) {

  /* we have to initialize the state */
  hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
              blockIdx.x, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              	&states[blockIdx.x]);
}

int main(int argc, char const *argv[])
{
	/* CUDA's random number library uses hiprandState_t to keep track of the seed value
     we will store a random state for every thread  */
	hiprandState_t* states;
  	/* allocate space on the GPU for the random states */
	HANDLE_ERROR( hipMalloc((void**) &states, POP_SIZE * sizeof(hiprandState_t) ) );
  	/* invoke the GPU to initialize all of the random states */
	init<<<POP_SIZE, 1>>>(time(0), states);

	// solutions[POPULATION_SIZE][SOLUTION_SIZE]
	// [p0v0 p0v1 p1v0 p1v1 p2v0 p2v1]
	double *dev_solutions_matrix; 
	double *dev_solutions_objectives;

	HANDLE_ERROR( hipMalloc((void**) &dev_solutions_matrix, SOLUTION_SIZE * POP_SIZE * sizeof(double) ) );
	HANDLE_ERROR( hipMalloc((void**) &dev_solutions_objectives, POP_SIZE * sizeof(double) ) );

	pso<<<POP_SIZE,1>>>(states, dev_solutions_matrix, dev_solutions_objectives);

	// hipDeviceSynchronize is used to allow printf inside device functions
	// http://stackoverflow.com/questions/19193468/why-do-we-need-cudadevicesynchronize-in-kernels-with-device-printf
	hipDeviceSynchronize();
	
	return 0;
}
