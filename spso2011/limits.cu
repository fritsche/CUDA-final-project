#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <math.h>
#include <assert.h>
#include <float.h>

#define CHK_CUDA(e) {if (e != hipSuccess) {fprintf(stderr,"Error: %s\n", hipGetErrorString(e)); exit(-1);}}

/* from wikipedia page, for machine epsilon calculation */
/* assumes mantissa in final bits */
__device__ double machine_eps_dbl() {
    typedef union {
        long long i64;
        double d64;
    } dbl_64;

    dbl_64 s;

    s.d64 = 1.;
    s.i64++;
    return (s.d64 - 1.);
}

__device__ float machine_eps_flt() {
    typedef union {
        int i32;
        float f32;
    } flt_32;

    flt_32 s;

    s.f32 = 1.;
    s.i32++;
    return (s.f32 - 1.);
}

#define EPS 0
#define MIN 1
#define MAX 2

__global__ void calc_consts(float *fvals, double *dvals) {

    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i==0) {
        fvals[EPS] = machine_eps_flt();
        dvals[EPS]= machine_eps_dbl();

        float xf, oldxf;
        double xd, oldxd; 

        xf = 2.; oldxf = 1.;
        xd = 2.; oldxd = 1.;

        /* double until overflow */
        /* Note that real fmax is somewhere between xf and oldxf */
        while (!isinf(xf))  {
            oldxf *= 2.;
            xf *= 2.;
        }

        while (!isinf(xd))  {
            oldxd *= 2.;
            xd *= 2.;
        }

        dvals[MAX] = oldxd;
        fvals[MAX] = oldxf;

        /* half until overflow */
        /* Note that real fmin is somewhere between xf and oldxf */
        xf = 1.; oldxf = 2.;
        xd = 1.; oldxd = 2.;

        while (xf != 0.)  {
            oldxf /= 2.;
            xf /= 2.;
        }

        while (xd != 0.)  {
            oldxd /= 2.;
            xd /= 2.;
        }

        dvals[MIN] = oldxd;
        fvals[MIN] = oldxf;

    }
    return;
}

int main(int argc, char **argv) {
    float  fvals[3];
    double dvals[3];
    float  *fvals_d;
    double *dvals_d;

    CHK_CUDA( hipMalloc(&fvals_d, 3*sizeof(float)) );
    CHK_CUDA( hipMalloc(&dvals_d, 3*sizeof(double)) );

    calc_consts<<<1,32>>>(fvals_d, dvals_d);

    CHK_CUDA( hipMemcpy(fvals, fvals_d, 3*sizeof(float), hipMemcpyDeviceToHost) );
    CHK_CUDA( hipMemcpy(dvals, dvals_d, 3*sizeof(double), hipMemcpyDeviceToHost) );

    CHK_CUDA( hipFree(fvals_d) );
    CHK_CUDA( hipFree(dvals_d) );

    printf("Single machine epsilon:\n");
    printf("CUDA = %g, CPU = %g\n", fvals[EPS], FLT_EPSILON);
    printf("Single min value (CUDA - approx):\n");
    printf("CUDA = %g, CPU = %g\n", fvals[MIN], FLT_MIN);
    printf("Single max value (CUDA - approx):\n");
    printf("CUDA = %g, CPU = %g\n", fvals[MAX], FLT_MAX);

    printf("\nDouble machine epsilon:\n");
    printf("CUDA = %lg, CPU = %lg\n", dvals[EPS], DBL_EPSILON);
    printf("Double min value (CUDA - approx):\n");
    printf("CUDA = %lg, CPU = %lg\n", dvals[MIN], DBL_MIN);
    printf("Double max value (CUDA - approx):\n");
    printf("CUDA = %lg, CPU = %lg\n", dvals[MAX], DBL_MAX);

    return 0;
}